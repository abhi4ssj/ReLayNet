
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void my_pooling(float* const out, float const* const data, size_t const H, size_t const W, size_t const C, size_t const N, size_t const stride) 
{
	//channel
	const size_t d = blockIdx.x;
	//image number	
	const size_t n = blockIdx.y;

	// spatial dimensions of the output array
	const size_t H_out = (size_t) ceilf(H/stride);
	const size_t W_out = (size_t) ceilf(W/stride);
	const size_t R = H_out * W_out;
	const size_t regTi = (size_t) ceilf(R / blockDim.x);


	printf("regTi=%d\n", regTi);

	// for each of the regions assigned to the current thread
	for(size_t reg=regTi*threadIdx.x; reg < min((int)regTi*(threadIdx.x+1),(int) R); reg++)
	{
		//get the base (v,u) positions of the input Image
		size_t vIN = stride * ( reg % H_out ); 
		size_t uIN = stride * floorf( reg / H_out );

		// -inf
		float max_ =  __int_as_float(0xff800000);
		for(size_t v_ = vIN; v_ < min(int(vIN + stride), int(H)); v_++)
		{
			for(size_t u_ = uIN; u_ < min(int(uIN + stride), int(W)); u_++)
			{
				size_t indIN = n*(H*W*C) + d*(H*W) + uIN*(H) + vIN;
				if(data[indIN] > max_)
					max_ = data[indIN];
			}
		}

		//assign result to output
		size_t vOUT = reg % H_out;
		size_t uOUT = floorf(reg / H_out);
		size_t indOUT = n*(H_out*W_out*C) + d*(H_out*W_out) + uOUT*(H_out) + vOUT;
		out[indOUT] = max_;
	}
}

__global__ void my_poolingIndices(float* const outMax, int* const outIndices, float const* const data, size_t const H, size_t const W, size_t const C, size_t const N, size_t const stride) 
{
	//channel
	const size_t d = blockIdx.x;
	//image number	
	const size_t n = blockIdx.y;

	// spatial dimensions of the output array
	const size_t H_out = (size_t) ceil(double(H)/double(stride));
	const size_t W_out = (size_t) ceil(double(W)/double(stride));
	const size_t R = H_out * W_out;
	const size_t regTi = (size_t) ceil(double(R) / double(blockDim.x));

	// for each of the regions assigned to the current thread
	for(size_t reg=regTi*threadIdx.x; reg < min((int)regTi*(threadIdx.x+1),(int) R); reg++)
	{
		//get the base (v,u) positions of the input Image
		size_t vIN = stride * ( reg % H_out ); 
		size_t uIN = (size_t)stride * floor( double(reg) / double(H_out) );

		// -inf
		float max_ =  __int_as_float(0xff800000);
		int maxIdx_ = -1;
		for(size_t v_ = vIN; v_ < min(int(vIN + stride), int(H)); v_++)
		{
			for(size_t u_ = uIN; u_ < min(int(uIN + stride), int(W)); u_++)
			{
				size_t indIN = n*(H*W*C) + d*(H*W) + u_*(H) + v_;
				if(data[indIN] > max_)
				{
					max_ = data[indIN];
					maxIdx_ = indIN;
				}
			}
		}

		//assign result to output
		size_t vOUT = reg % H_out;
		size_t uOUT = (size_t)floor(double(reg) / double(H_out));

		size_t indOUT = n*(H_out*W_out*C) + d*(H_out*W_out) + uOUT*(H_out) + vOUT;
		outMax[indOUT] = max_;
		outIndices[indOUT] = maxIdx_;
	}
}


__global__ void test(int* const deb, float const* const data)
{
		size_t const stride = 2;
		size_t reg = 93;
		const size_t H_out = (size_t) ceil(double(360)/double(stride));
		size_t uIN = (size_t)stride * floor( double(reg) / double(H_out) );

		printf("uIN = %d, H_out = %d\n", uIN, H_out);

		deb[0] = uIN;
		deb[1] = H_out;
}

